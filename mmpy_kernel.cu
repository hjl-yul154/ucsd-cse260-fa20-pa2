#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    //local shared storage
    int TY = blockDim.y;
    int TX = blockDim.x;
    int TW = blockDim.x;

    __shared__ _DOUBLE_ As[TW][TW];
    __shared__ _DOUBLE_ Bs[TW][TW];
    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = blockIdx.x;
    int I = by*TW + ty; int J= bx*TW + tx;
    double Cij = 0;
    for (int kk=0; kk<N/TW; kk++){
        As[ty][tx] = A[I*N + kk*TW+tx];
        Bs[ty][tx] = B[(kk*TW+ty)*N + J];
        __syncthreads();
        for (int k=0; k<TW; k++)
            Cij+= As[ty][k] * Bs[k][tx];
        __syncthreads();
    }
    C[I*N + J] = Cij;
}

__global__ void matMul_old(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _DOUBLE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _DOUBLE_ a = A[I * N + k];
            _DOUBLE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}
