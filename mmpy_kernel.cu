#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

#define BLOCK_SIZE_M 96
#define BLOCK_SIZE_N 64
#define BLOCK_SIZE_K 32
#if BLOCK_SIZE_M % BLOCKDIM_Y || BLOCK_SIZE_K % BLOCKDIM_X
#error Use thread block to load block of A
#endif
#if BLOCK_SIZE_K % BLOCKDIM_Y || BLOCK_SIZE_N % BLOCKDIM_X
#error Use thread block to load block of B
#endif
#if BLOCK_SIZE_M % BLOCKDIM_Y || BLOCK_SIZE_N % BLOCKDIM_X
#error Use thread block to compute block of C
#endif
// Number of sub-block of C for each thread
#define X_SUB (BLOCK_SIZE_N / BLOCKDIM_X)
#define Y_SUB (BLOCK_SIZE_M / BLOCKDIM_Y)

#define MAT(mat, N, i, j) (mat[(i)*N + (j)])
#define MAT_PADDED(mat, N, i, j) ((i) < N && (j) < N ? MAT(mat, N, i, j) : 0)
#define A_ELEMENT(i, j) MAT_PADDED(A, N, i, j)
#define B_ELEMENT(i, j) MAT_PADDED(B, N, i, j)
#define C_ELEMENT(i, j) MAT(C, N, i, j)

#define get_mat(mat,N,i,j)((i<N)&&(j<N)?mat[i*N+j]:0)

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
    __shared__ _DOUBLE_ Ab[BM][BK];
    __shared__ _DOUBLE_ Bb[BK][BN];
    _DOUBLE_ frag_a[TM];
    _DOUBLE_ frag_b[TN];
    _DOUBLE_ Cb[TM][TN]={0};


    int ty = threadIdx.y, tx=threadIdx.x;
    int by = blockIdx.y, bx=blockIdx.x;

    int I =  by*BM + ty;
    int J =  bx*BN + tx;

    int I0 = by * BLOCK_SIZE_M;
    int J0 = bx * BLOCK_SIZE_N;

#pragma unroll
    for (int K = 0; K < N; K += BLOCK_SIZE_K) {
#pragma unroll
        for (int i = 0; i < BLOCK_SIZE_M; i += BLOCKDIM_Y) {
#pragma unroll
            for (int j = 0; j < BLOCK_SIZE_K; j += BLOCKDIM_X) {
                Ab[ty+i][tx+j]=get_mat(A,N,I+i,K+tx+j);
            }
        }

#pragma unroll
        for (int i = 0; i < BLOCK_SIZE_K; i += BLOCKDIM_Y) {
#pragma unroll
            for (int j = 0; j < BLOCK_SIZE_N; j += BLOCKDIM_X) {
                Bb[ty+i][tx+j]=get_mat(B,N,K+ty+i,J+j);
            }
        }
        __syncthreads();
        #pragma unroll
        for (int k=0;k<BK;++k){
            #pragma unroll
            for (int i=0;i<TM;++i){
                frag_a[i]=Ab[ty+BY*i][k];
            }
            #pragma unroll
            for (int j=0;j<TN;++j){
                frag_b[j]=Bb[k][tx+BX*j];
            }

            #pragma unroll
            for (int i=0;i<TM;++i){
                #pragma unroll
                for (int j=0;j<TN;++j){
                    Cb[i][j]+=frag_a[i]*frag_b[j];
                }
            }
        }
        __syncthreads();
    }
    #pragma unroll
    for(int i=0;i<TM;++i){
        #pragma unroll
        for(int j=0;j<TN;++j){
            if(I+i*BY<N && J+j*BX<N){
                C[(I+BY*i)*N+J+BX*j]=Cb[i][j];
            }
        }
    }

}


#define TW 16
__global__ void matMul_shared(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    //local shared storage
    int TY = blockDim.y;
    int TX = blockDim.x;
//     int TW = blockDim.x;

    __shared__ _DOUBLE_ As[TW][TW];
    __shared__ _DOUBLE_ Bs[TW][TW];
    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = blockIdx.x;
    int I = by*TW + ty; int J= bx*TW + tx;
    double Cij = 0;
    for (int kk=0; kk<N/TW; ++kk){
        As[ty][tx] = A[I*N + kk*TW+tx];
        Bs[ty][tx] = B[(kk*TW+ty)*N + J];
        __syncthreads();
        for (int k=0; k<TW; ++k)
            Cij+= As[ty][k] * Bs[k][tx];
        __syncthreads();
    }
    C[I*N + J] = Cij;
}

__global__ void matMul_naive(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _DOUBLE_ _c = 0;
        for (unsigned int k = 0; k < N; ++k) {
            _DOUBLE_ a = A[I * N + k];
            _DOUBLE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}
